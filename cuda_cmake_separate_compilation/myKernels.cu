#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include "myKernels.h"

// extern "C" is required to make .cpp files recognize and use the kernel
extern "C" __global__ void reciprocalKernel(float *data, unsigned vectorSize) {
	unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < vectorSize)
		data[idx] = 1.0/data[idx];
}

float *gpuReciprocal(float *data, unsigned size)
{
	float *rc = new float[size];
	float *gpuData;


	hipMalloc((void **)&gpuData, sizeof(float)*size);
	hipMemcpy(gpuData, data, sizeof(float)*size, hipMemcpyHostToDevice);

	static const int BLOCK_SIZE = 256;
	const int blockCount = (size+BLOCK_SIZE-1)/BLOCK_SIZE;
	reciprocalKernel<<<blockCount, BLOCK_SIZE>>> (gpuData, size);

	//CUDA_CHECK_RETURN(hipMemcpy(rc, gpuData, sizeof(float)*size, hipMemcpyDeviceToHost));
	//CUDA_CHECK_RETURN(hipFree(gpuData));
	hipMemcpy(rc, gpuData, sizeof(float)*size, hipMemcpyDeviceToHost);
	hipFree(gpuData);

	return rc;
}

float *cpuReciprocal(float *data, unsigned size)
{
	float *rc = new float[size];
	for (unsigned cnt = 0; cnt < size; ++cnt) rc[cnt] = 1.0/data[cnt];
	return rc;
}
void initialize(float *data, unsigned size)
{
	for (unsigned i = 0; i < size; ++i)
		data[i] = .5*(i+1);
}


